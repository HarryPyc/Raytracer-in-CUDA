#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <stdio.h>


void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();

	if (hipSuccess != err)
	{
		fprintf(stderr, "CUDA error: %s: %s. \n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


__global__ void render(uchar4 *pos, int width, int height) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	if ((i >= width) || (j >= height)) 
		return;

	int index = j * width + i;

	unsigned char r = int(float(i) / width * 255.99) & 0xff;
	unsigned char g = int(float(j) / height * 255.99) & 0xff;
	unsigned char b = (70) & 0xff;

	pos[index].w = 0;
	pos[index].x = r;
	pos[index].y = g;
	pos[index].z = b;
}

extern "C" void launch_kernel(uchar4* pos, unsigned int w, unsigned int h) {

	int tx = 8;
	int ty = 8;

	dim3 blocks(w / tx + 1, h / ty + 1);
	dim3 threads(tx, ty);
	render <<<blocks, threads >>> (pos, w, h);


	hipDeviceSynchronize();
	checkCUDAError("kernel failed!");
}

